#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <thrust/transform.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <vector>
#include <iostream>
#include <ctime>
#include <cmath>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>

#define GLEW_STATIC
#include <GL/glew.h>

#include <GLFW/glfw3.h>
#include "Shader.h"

#include "tiny_obj_loader.h"

#include "matrix.h"
#include "triangle.h"
#include "multi_rasterizer.h"
#include "index_to_clipspace_functor.h"

using namespace thrust;

const int N = 512;
const int WIDTH = N, HEIGHT = N;

struct rasterize_functor {
	const kp::index_to_clipspace_functor index_to_clipspace;
	const kp::multi_rasterizer rasterizer;

	__host__ __device__
		rasterize_functor(const kp::index_to_clipspace_functor index_to_clipspace, const kp::multi_rasterizer rasterizer)
		: index_to_clipspace(index_to_clipspace), rasterizer(rasterizer) {}

	__host__ __device__
		tuple<float, float, float, float, int> operator()(int i) {
			return rasterizer(index_to_clipspace(i));
		}
};

struct std_scene {
	std::vector<float> vertices_x;
	std::vector<float> vertices_y;
	std::vector<float> vertices_z;
	std::vector<unsigned int> triangles_a;
	std::vector<unsigned int> triangles_b;
	std::vector<unsigned int> triangles_c;
};

float quadf(float value, float max) {
	return std::cosf(((value - (max / 2)) / (max / 2)) * M_PI) / 2 + 0.5f;
}

std_scene generate_cosine_quad() {
	std_scene scene;

	int imax = 91;
	int jmax = 91;

	for (int i = 0; i < imax; i++) {
		for (int j = 0; j < jmax; j++) {
			scene.vertices_x.push_back(((float)j / (jmax - 1)) * 2.f - 1.f);
			scene.vertices_y.push_back(((float)i / (imax - 1)) * (-2.f) + 1.f);
			scene.vertices_z.push_back(quadf(i, imax) * quadf(j, jmax));
		}
	}

	for (int i = 0; i < imax - 1; i++) {
		for (int j = 0; j < jmax - 1; j++) {
			scene.triangles_a.push_back(i*jmax + j);
			scene.triangles_b.push_back((i + 1)*imax + j);
			scene.triangles_c.push_back(i*imax + j + 1);

			scene.triangles_a.push_back((i + 1)*jmax + (j + 1));
			scene.triangles_b.push_back(i*imax + j + 1);
			scene.triangles_c.push_back((i + 1)*imax + j);
		}
	}

	return scene;
}

std_scene load_scene() {
	std::string inputfile = "scenes/cube.obj";
	std::string err;
	std::vector<tinyobj::shape_t> shapes;
	std::vector<tinyobj::material_t> materials;

	if (!tinyobj::LoadObj(shapes, materials, err, inputfile.c_str())) {
		std::cerr << err << std::endl;
		exit(1);
	}

	std::cout << "# of shapes    : " << shapes.size() << std::endl;
	std::cout << "# of materials : " << materials.size() << std::endl;

	std_scene scene;
	auto max_vertex_value = *std::max_element(shapes[0].mesh.positions.begin(), shapes[0].mesh.positions.end());
	auto vertex_factor = 1.f / max_vertex_value;
	std::cout << "Max vertex value :" << max_vertex_value << std::endl;

	for (size_t i = 0; i < shapes.size(); i++) {
		auto shape = shapes[i];
		auto offset = scene.triangles_a.size();

		for (size_t j = 0; j < shape.mesh.indices.size(); j += 3) {
			scene.triangles_a.push_back(offset + shape.mesh.indices[j + 0]);
			scene.triangles_b.push_back(offset + shape.mesh.indices[j + 1]);
			scene.triangles_c.push_back(offset + shape.mesh.indices[j + 2]);
		}

		for (size_t j = 0; j < shape.mesh.positions.size(); j += 3) {
			scene.vertices_x.push_back(shape.mesh.positions[j + 0] / 3/* * vertex_factor - 0.5f*/);
			scene.vertices_y.push_back(shape.mesh.positions[j + 1] / 3/* * vertex_factor - 0.5f*/);
			scene.vertices_z.push_back(shape.mesh.positions[j + 2] / 3/* * vertex_factor - 0.5f*/);
		}
	}

	return scene;
}

void generate_image2(unsigned char* image, std_scene scene) {
	std::cout << "Number of triangles: " << scene.triangles_a.size() << std::endl;
	auto size = WIDTH * HEIGHT;
	device_vector<float> screen_x(size), screen_y(size), screen_z(size), screen_depth(size);
	device_vector<int> screen_triangles(size);
	counting_iterator<int> begin(0);
	counting_iterator<int> end(size);
	device_vector<int> indices(size);
	sequence(indices.begin(), indices.end());

	auto t_begin = std::clock();

	//std::vector<float> std_vertices_x{ -1.0f, 0.66f, 0.0f, 1.0f, -0.75f, 0.0f, -1.0f, 1.0f };
	//std::vector<float> std_vertices_y{ -0.75f, -1.0f, 1.0f, 1.0f, 0.75f, -1.0f, 0.0f, 0.0f };
	//std::vector<float> std_vertices_z{ 0.5f, 0.5f, -0.25f, 0.5f, -0.75f, -0.75f, -0.55f, 1.0f };

	//// Indices for corners A, B and C of triangles to be rasterized
	//std::vector<unsigned int> std_triangles_a{ 5, 0, 2, 0 };
	//std::vector<unsigned int> std_triangles_b{ 7, 1, 1, 2 };
	//std::vector<unsigned int> std_triangles_c{ 6, 2, 3, 4 };

	// Copy vertices and triangles to GPU
	device_vector<float> vertices_x = scene.vertices_x;
	device_vector<float> vertices_y = scene.vertices_y;
	device_vector<float> vertices_z = scene.vertices_z;
	device_vector<unsigned int> triangles_a = scene.triangles_a;
	device_vector<unsigned int> triangles_b = scene.triangles_b;
	device_vector<unsigned int> triangles_c = scene.triangles_c;

	//for (size_t i = 0; i < scene.vertices_x.size(); i++) {
	//	std::cout << "x:\t" << scene.vertices_x[i] << "y:\t" << scene.vertices_y[i] << "z:\t" << scene.vertices_z[i] << std::endl;
	//}

	//for (size_t i = 0; i < scene.triangles_a.size(); i++) {
	//	std::cout << "a:\t" << scene.triangles_a[i] << "b:\t" << scene.triangles_b[i] << "c:\t" << scene.triangles_c[i] << std::endl;
	//}

	const kp::index_to_clipspace_functor index_to_clipspace(WIDTH, HEIGHT);
	const kp::multi_rasterizer rasterizer(
		triangles_a.size(),
		vertices_x.data(),
		vertices_y.data(),
		vertices_z.data(),
		triangles_a.data(),
		triangles_b.data(),
		triangles_c.data());

	auto screen_begin = make_tuple(screen_x.begin(), screen_y.begin(), screen_z.begin(), screen_depth.begin(), screen_triangles.begin());
	auto screen_end = make_tuple(screen_x.end(), screen_y.end(), screen_z.end(), screen_depth.end(), screen_triangles.end());

	transform(indices.begin(), indices.end(), make_zip_iterator(screen_begin), rasterize_functor(index_to_clipspace, rasterizer));
	hipDeviceSynchronize();

	auto t_end = std::clock();
	auto elapsed_secs = double(t_end - t_begin) / CLOCKS_PER_SEC;
	std::cout << "Time elapsed: " << elapsed_secs*1000.0 << "ms" << std::endl;

	host_vector<float> host_x(size), host_y(size), host_z(size), host_depth(size);
	host_vector<int> host_triangles(size);
	auto host_begin = make_tuple(host_x.begin(), host_y.begin(), host_z.begin(), host_depth.begin(), host_triangles.begin());
	copy(make_zip_iterator(screen_begin), make_zip_iterator(screen_end), make_zip_iterator(host_begin));

	//auto factor = 255 / std_triangles_a.size();
	for (int i = 0; i < size; i++) {
		image[i * 3 + 0] = (unsigned char)((host_depth[i] * 0.5f + 0.5f) * 255);
		image[i * 3 + 1] = (unsigned char)((host_depth[i] * 0.5f + 0.5f) * 255);
		image[i * 3 + 2] = (unsigned char)((host_depth[i] * 0.5f + 0.5f) * 255);
	}
	//for (int i = 0; i < size; i++) {
	//	image[i * 3 + 0] = (unsigned char)((host_x[i]) * 255);
	//	image[i * 3 + 1] = (unsigned char)((host_y[i]) * 255);
	//	image[i * 3 + 2] = (unsigned char)((host_z[i]) * 255);
	//}
}

// Function prototypes
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode);

// Window dimensions

int main() {
	// Init GLFW
	glfwInit();
	// Set all the required options for GLFW
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);

	// Create a GLFWwindow object that we can use for GLFW's functions
	GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "02507 CUDA Rasterizer", nullptr, nullptr);
	glfwMakeContextCurrent(window);

	// Set the required callback functions
	glfwSetKeyCallback(window, key_callback);


	// Set this to true so GLEW knows to use a modern approach to retrieving function pointers and extensions
	glewExperimental = GL_TRUE;
	// Initialize GLEW to setup the OpenGL Function pointers
	glewInit();

	// Define the viewport dimensions
	glViewport(0, 0, WIDTH, HEIGHT);


	// Build and compile our shader program
	Shader ourShader("textures.vert", "textures.frag");


	// Set up vertex data (and buffer(s)) and attribute pointers
	GLfloat vertices[] = {
		// Positions          // Colors           // Texture Coords
		1.0f, 1.0f, 0.0f, 1.0f, 0.0f, 0.0f, 1.0f, 1.0f, // Top Right
		1.0f, -1.0f, 0.0f, 0.0f, 1.0f, 0.0f, 1.0f, 0.0f, // Bottom Right
		-1.0f, -1.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f, // Bottom Left
		-1.0f, 1.0f, 0.0f, 1.0f, 1.0f, 0.0f, 0.0f, 1.0f  // Top Left 
	};
	GLuint indices[] = {  // Note that we start from 0!
		0, 1, 3, // First Triangle
		1, 2, 3  // Second Triangle
	};
	GLuint VBO, VAO, EBO;
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);
	glGenBuffers(1, &EBO);

	glBindVertexArray(VAO);

	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// Position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(GLfloat), (GLvoid*)0);
	glEnableVertexAttribArray(0);
	// Color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(GLfloat), (GLvoid*)(3 * sizeof(GLfloat)));
	glEnableVertexAttribArray(1);
	// TexCoord attribute
	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(GLfloat), (GLvoid*)(6 * sizeof(GLfloat)));
	glEnableVertexAttribArray(2);

	glBindVertexArray(0); // Unbind VAO


	// Load and create a texture 
	GLuint texture;
	glGenTextures(1, &texture);
	glBindTexture(GL_TEXTURE_2D, texture); // All upcoming GL_TEXTURE_2D operations now have effect on this texture object
	// Set the texture wrapping parameters
	//glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	// Set texture wrapping to GL_REPEAT (usually basic wrapping method)
	//glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	// Set texture filtering parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// Load image, create texture
	int widths = WIDTH;
	int heights = HEIGHT;
	unsigned char* image = new unsigned char[widths*heights * 3];

	auto scene = generate_cosine_quad();
	generate_image2(image, scene);
	//generate_image2(image, scene);
	//generate_image2(image, scene);
	//generate_image2(image, scene);
	//generate_image2(image, scene);

	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, widths, heights, 0, GL_RGB, GL_UNSIGNED_BYTE, image);
	//glGenerateMipmap(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, 0); // Unbind texture when done, so we won't accidentily mess up our texture.


	// Game loop
	while (!glfwWindowShouldClose(window))
	{
		// Check if any events have been activiated (key pressed, mouse moved etc.) and call corresponding response functions
		glfwPollEvents();

		// Render
		// Clear the colorbuffer
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);


		// Bind Texture
		glBindTexture(GL_TEXTURE_2D, texture);

		// Activate shader
		ourShader.Use();

		// Draw container
		glBindVertexArray(VAO);
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
		glBindVertexArray(0);

		// Swap the screen buffers
		glfwSwapBuffers(window);
	}
	// Properly de-allocate all resources once they've outlived their purpose
	glDeleteVertexArrays(1, &VAO);
	glDeleteBuffers(1, &VBO);
	glDeleteBuffers(1, &EBO);
	// Terminate GLFW, clearing any resources allocated by GLFW.
	glfwTerminate();
	return 0;
}

// Is called whenever a key is pressed/released via GLFW
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode)
{
	if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
		glfwSetWindowShouldClose(window, GL_TRUE);
}